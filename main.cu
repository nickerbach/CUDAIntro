#include "hip/hip_runtime.h"
#include "book.h"
#include <stdio.h>
__global__ void assignIdentifier(int *r)
{
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	r[c] = c;
}
int main(int argc, char ** argv )
{   
    int test = atoi(argv[1]);
    if(argc == 1)
    {
        test = 512;
    }
    int *dev_a;
    int a[test];
    HANDLE_ERROR(hipMalloc( (void**)&dev_a, test * sizeof(int)) );
    assignIdentifier<<<1,test>>>(dev_a);
    HANDLE_ERROR(hipMemcpy(a, dev_a, test * sizeof(int), hipMemcpyDeviceToHost));
    for(int i = 0; i< test; i++)
    {
	        printf( "%d\n", a[i]);
    }
    hipFree(dev_a);
    return 0;
}
