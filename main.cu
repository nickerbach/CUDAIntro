#include "hip/hip_runtime.h"
#include "book.h"//connects the program to the book.h file
#include <stdio.h>//import needed to use atoi
__global__ void assignIdentifier(int *r)//kernel that gets the thread ID then saves the value
{
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	r[c] = c;
}
int main(int argc, char ** argv )
{   
    printf("%d\n",argc);
    int test;//gets the command line argument of how many threads a user wants to use, casts to int
    if(argc < 2)//if no command line entry is put in, defaults to 512 threads
    {
        test = 512;
    }
    else
    {
	test = atoi(argv[1]);//sets number of threads to the user input
     }
    int *dev_a;//pointer to the spot in memory we will use
    int a[test];//array we will use to store the ids before printing
    HANDLE_ERROR(hipMalloc( (void**)&dev_a, test * sizeof(int)) );//creates memory space on the GPU
    assignIdentifier<<<1,test>>>(dev_a);//uses kernel to store ids for each thread
    HANDLE_ERROR(hipMemcpy(a, dev_a, test * sizeof(int), hipMemcpyDeviceToHost));//copies our array from GPU to CPU
    for(int i = 0; i< test; i++)//prints each id back to the user
    {
	        printf( "%d\n", a[i]);
    }
    hipFree(dev_a);//frees up memory space that we used now that we are done
    return 0;
}
